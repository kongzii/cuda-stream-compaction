#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Config/Config.hpp"

#include "data.hpp"
#include "utils.hpp"

// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html

static void handleCUDAError(
        hipError_t error,        // error code
        const char *file,         // file within error was generated
        int line)                 // line where error occurs
{
    if (error != hipSuccess) {    // any error -> display error message and terminate application
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CHECK_ERROR(error) ( handleCUDAError( error, __FILE__, __LINE__ ) )

#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

// Filtering

__global__ void cuda_filter(Data *input, int *filter, int elements_per_block, int interval_from, int interval_to) {
    int bid = blockIdx.x;
    int block_offset = bid * elements_per_block;

    int tid = threadIdx.x;

    filter[block_offset + tid] = FILTER(input[block_offset + tid], interval_from, interval_to);
}

void perform_filter(Data *input, int *filter, int threads_per_block, int size, int interval_From, int interval_to) {
    int n_blocks = size / threads_per_block;

    size_t input_size = size * sizeof(Data);
    size_t filter_size = size * sizeof(int);

    Data *input_cuda;
    int *filter_cuda;

    CHECK_ERROR(hipMalloc((void **) &input_cuda, input_size));
    CHECK_ERROR(hipMalloc((void **) &filter_cuda, filter_size));

    CHECK_ERROR(hipMemcpy(input_cuda, input, input_size, hipMemcpyHostToDevice));

    cuda_filter <<< n_blocks, threads_per_block >>> (input_cuda, filter_cuda, threads_per_block, interval_From, interval_to);

    int remains = size - n_blocks * threads_per_block;

    if (remains > 0) {
        cuda_filter <<< 1, remains >>> (&(input_cuda[size - remains]), &(filter_cuda[size - remains]), remains, interval_From, interval_to);
    }

    CHECK_ERROR(hipMemcpy(filter, filter_cuda, filter_size, hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(input_cuda));
    CHECK_ERROR(hipFree(filter_cuda));
}

// Adding

__global__ void add(int *output, int length, int *n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int block_offset = bid * length;

    output[block_offset + tid] += n[bid];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int block_offset = bid * length;

    output[block_offset + tid] += n1[bid] + n2[bid];
}

// Scanning

__global__ void cuda_scan_small(int *filter, int *scan, int size, int size_pot) {
    extern __shared__ int sh_data[];

    int thid = threadIdx.x;

    int ai = thid;
    int bi = thid + (size / 2);

    int bank_offset_a = CONFLICT_FREE_OFFSET(ai);
    int bank_offset_b = CONFLICT_FREE_OFFSET(bi);

    if (thid < size) {
        sh_data[ai + bank_offset_a] = filter[ai];
        sh_data[bi + bank_offset_b] = filter[bi];
    } else {
        sh_data[ai + bank_offset_a] = 0;
        sh_data[bi + bank_offset_b] = 0;
    }

    int offset = 1;

    for (int d = size_pot >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            sh_data[bi] += sh_data[ai];
        }

        offset *= 2;
    }

    if (thid == 0) {
        sh_data[size_pot - 1 + CONFLICT_FREE_OFFSET(size_pot - 1)] = 0;
    }

    for (int d = 1; d < size_pot; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = sh_data[ai];
            sh_data[ai] = sh_data[bi];
            sh_data[bi] += t;
        }
    }

    __syncthreads();

    if (thid < size) {
        scan[ai] = sh_data[ai + bank_offset_a];
        scan[bi] = sh_data[bi + bank_offset_b];
    }
}

__global__ void cuda_scan_large(int *filter, int *scan, int size, int *sums) {
    extern __shared__ int sh_data[];

    int bid = blockIdx.x;
    int thid = threadIdx.x;
    int block_offset = bid * size;

    int ai = thid;
    int bi = thid + (size / 2);

    int bank_offset_a = CONFLICT_FREE_OFFSET(ai);
    int bank_offset_b = CONFLICT_FREE_OFFSET(bi);

    sh_data[ai + bank_offset_a] = filter[block_offset + ai];
    sh_data[bi + bank_offset_b] = filter[block_offset + bi];

    int offset = 1;

    for (int d = size >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            sh_data[bi] += sh_data[ai];
        }

        offset *= 2;
    }

    __syncthreads();

    if (thid == 0) {
        sums[bid] = sh_data[size - 1 + CONFLICT_FREE_OFFSET(size - 1)];
        sh_data[size - 1 + CONFLICT_FREE_OFFSET(size - 1)] = 0;
    }

    for (int d = 1; d < size; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = sh_data[ai];
            sh_data[ai] = sh_data[bi];
            sh_data[bi] += t;
        }
    }

    __syncthreads();

    scan[block_offset + ai] = sh_data[ai + bank_offset_a];
    scan[block_offset + bi] = sh_data[bi + bank_offset_b];
}

// Only declaration, because of scan_large <-> scan_large_even
void scan_large(int *filter_cuda, int *scan_cuda, int size, int elements_per_block, int threads_per_block);

void scan_small(int *filter_cuda, int *scan_cuda, int size) {
    int pot = next_power_of_two(size);

    cuda_scan_small <<< 1, (size + 1) / 2, 2 * pot * sizeof(int) >>> (filter_cuda, scan_cuda, size, pot);
}

void scan_large_even(int *filter_cuda, int *scan_cuda, int size, int elements_per_block, int threads_per_block) {
    int n_blocks = size / elements_per_block;

    size_t int_size = elements_per_block * sizeof(int);

    int *sums;
    int *incr;

    CHECK_ERROR(hipMalloc((void **) &sums, n_blocks * sizeof(int)));
    CHECK_ERROR(hipMalloc((void **) &incr, n_blocks * sizeof(int)));

    cuda_scan_large <<< n_blocks, threads_per_block, 2 * int_size >>> (filter_cuda, scan_cuda, elements_per_block, sums);

    int sums_threads = (n_blocks + 1) / 2;

    if (sums_threads > threads_per_block) {
        scan_large(sums, incr, n_blocks, elements_per_block, threads_per_block);
    } else {
        scan_small(sums, incr, n_blocks);
    }

    add <<< n_blocks, elements_per_block >>> (scan_cuda, elements_per_block, incr);

    CHECK_ERROR( hipFree(incr) );
    CHECK_ERROR( hipFree(sums) );
}

void scan_large(int *filter_cuda, int *scan_cuda, int size, int elements_per_block, int threads_per_block) {
    int remainder = size % elements_per_block;

    if (remainder == 0) {
        scan_large_even(filter_cuda, scan_cuda, size, elements_per_block, threads_per_block);
    }

    else {
        int size_multiple = size - remainder;

        scan_large_even(filter_cuda, scan_cuda, size_multiple, elements_per_block, threads_per_block);

        int *start_of_output = &(scan_cuda[size_multiple]);

        scan_small(&(filter_cuda[size_multiple]), start_of_output, remainder);

        add <<< 1, remainder >>> (start_of_output, remainder, &(filter_cuda[size_multiple - 1]), &(scan_cuda[size_multiple - 1]));
    }
}

void perform_scan(int *filter, int *scan, int threads_per_block, int size) {
    int elements_per_block = 2 * threads_per_block;

    int *filter_cuda;
    int *scan_cuda;

    size_t int_size = size * sizeof(int);

    CHECK_ERROR(hipMalloc((void **) &filter_cuda, int_size));
    CHECK_ERROR(hipMalloc((void **) &scan_cuda, int_size));

    CHECK_ERROR(hipMemcpy(filter_cuda, filter, int_size, hipMemcpyHostToDevice));

    if (size > elements_per_block) {
        scan_large(filter_cuda, scan_cuda, size, elements_per_block, threads_per_block);
    }

    else {
        scan_small(filter_cuda, scan_cuda, size);
    }

    CHECK_ERROR(hipMemcpy(scan, scan_cuda, int_size, hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(filter_cuda));
    CHECK_ERROR(hipFree(scan_cuda));
}

// Truncating

__global__ void cuda_truncate(Data *input, int *filter, int *scan, Data *filtered, int elements_per_block) {
    int bid = blockIdx.x;
    int block_offset = bid * elements_per_block;

    int tid = threadIdx.x;

    if (filter[block_offset + tid] == 1) {
        filtered[scan[block_offset + tid]] = input[block_offset + tid];
    }
}

void perform_truncate(Data *input, int *filter, int *scan, Data *filtered, int threads_per_block, int size, int new_size) {
    int n_blocks = size / threads_per_block;

    Data *input_cuda;
    int *filter_cuda;
    int *scan_cuda;
    Data *filtered_cuda;

    size_t data_size = size * sizeof(Data);
    size_t int_size = size * sizeof(int);
    size_t filtered_size = new_size * sizeof(Data);

    CHECK_ERROR(hipMalloc((void **) &input_cuda, data_size));
    CHECK_ERROR(hipMalloc((void **) &filter_cuda, int_size));
    CHECK_ERROR(hipMalloc((void **) &scan_cuda, int_size));
    CHECK_ERROR(hipMalloc((void **) &filtered_cuda, filtered_size));

    CHECK_ERROR(hipMemcpy(input_cuda, input, data_size, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(filter_cuda, filter, int_size, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(scan_cuda, scan, int_size, hipMemcpyHostToDevice));

    cuda_truncate <<< n_blocks, threads_per_block >>> (input_cuda, filter_cuda, scan_cuda, filtered_cuda, threads_per_block);

    int remains = size - n_blocks * threads_per_block;

    if (remains > 0) {
        cuda_truncate <<< 1, remains >>> (&(input_cuda[size - remains]), &(filter_cuda[size - remains]), &(scan_cuda[size - remains]), filtered_cuda, remains);
    }

    CHECK_ERROR(hipMemcpy(filtered, filtered_cuda, filtered_size, hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(input_cuda));
    CHECK_ERROR(hipFree(filter_cuda));
    CHECK_ERROR(hipFree(scan_cuda));
    CHECK_ERROR(hipFree(filtered_cuda));
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        EXIT("Usage: ./CSC path_to_config_file")
    }

    // Constants from config

    const auto config = Config(argv[1]);

    const auto SIZE = config.get_int("N");
    const auto THREADS_PER_BLOCK = config.get_int("THREADS_PER_BLOCK");

    if (THREADS_PER_BLOCK < 1) {
        EXIT("At least one THREADS_PER_BLOCK required")
    }

    const auto KEY_FROM = config.get_int("KEY_FROM");
    const auto KEY_TO = config.get_int("KEY_TO");

    const auto INTERVAL_FROM = config.get_int("INTERVAL_FROM");
    const auto INTERVAL_TO = config.get_int("INTERVAL_TO");

    // Generate input data

    Data *input;

    input = (Data *) malloc(SIZE * sizeof(Data));

    generate(input, SIZE, KEY_FROM, KEY_TO);

    // Perform filter

    int *filter;

    filter = (int *) malloc(SIZE * sizeof(int));

    perform_filter(input, filter, THREADS_PER_BLOCK, SIZE, INTERVAL_FROM, INTERVAL_TO);

    // Perform scan

    int *scan;

    scan = (int *) malloc(SIZE * sizeof(int));

    perform_scan(filter, scan, THREADS_PER_BLOCK, SIZE);

    // Get new size

    int filtered_size;

    if (scan[SIZE - 1] == 0 && !FILTER(input[SIZE - 1], INTERVAL_FROM, INTERVAL_TO)) {
        filtered_size = 0;
    }

    else if (scan[SIZE - 1] > 0 && FILTER(input[SIZE - 1], INTERVAL_FROM, INTERVAL_TO)) {
        filtered_size = scan[SIZE - 1] + 1;
    }

    else {
        filtered_size = scan[SIZE - 1];
    }

    // Perform truncation

    Data *filtered;

    filtered = (Data *) malloc(filtered_size * sizeof(Data));

    perform_truncate(input, filter, scan, filtered, THREADS_PER_BLOCK, SIZE, filtered_size);

    // Just print

    print("Input", input, SIZE);
    print("Filter", filter, SIZE);
    print("Scan", scan, SIZE);
    print("Final", filtered, filtered_size);

    std::cout << std::endl << "Filtered out " << SIZE - filtered_size << " items from " << SIZE << std::endl;

    // Free host memory

    free(input);
    free(filtered);
    free(filter);
    free(scan);
}